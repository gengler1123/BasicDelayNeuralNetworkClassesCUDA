
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <vector>
#include <ctime>
#include <iostream>

#include "kernels.cuh"
#include "edge.cuh"
#include "neuron.cuh"


		 int main()
{
	int numNeurons = 1000;
	int numExcit = 800;
	int T = 2000;
	int equilizationTime = 100;
	int transientTime = 300;
	int maxDelay = 15;

	/* CUDA Parameters */
	int numThreads = 512;

	/* Neurons */

	Neuron *h_neurons, *d_neurons;

	float *h_I, *d_I, *h_driven, *d_driven;
	bool *d_cf, *h_cf;

	float *d_inputs;

	float *d_v;

	h_neurons = new Neuron[numNeurons];
	h_I = new float[numNeurons*maxDelay];
	h_cf = new bool[numNeurons];
	h_driven = new float[numNeurons];

	float *h_drivenZero = new float[numNeurons];
	float *d_drivenZero;

	bool **SpikeTrainYard = new bool*[T];
	float **VoltageTrace = new float *[T];
	float **InputTrace = new float*[T];

	for (int i = 0; i < numNeurons; i++)
	{
		for (int j = 0; j < maxDelay; j++)
		{
			h_I[i*maxDelay + j] = 0;
		}

		if (i < numExcit)
		{
			h_neurons[i].setRegSpike();
		}
		else
		{
			h_neurons[i].setIntBurst();
		}

		h_cf[i] = false;

		if (i < 100)
		{
			h_driven[i] = 100;
			h_drivenZero[i] = 0;
		}
		else
		{
			h_driven[i] = 0;
			h_drivenZero[i] = 0;
		}
	}

	for (int t = 0; t < T; t++)
	{
		SpikeTrainYard[t] = new bool[numNeurons];
		VoltageTrace[t] = new float[numNeurons];
		InputTrace[t] = new float[numNeurons];
	}


	/* Edges */

	std::vector<Edge> h_edges; Edge *d_edges;

	std::mt19937 rd(time(NULL));
	std::uniform_real_distribution<float> dist(0.0, 1.0);
	std::uniform_int_distribution<int> intDist(1, maxDelay);

	int d;
	float w;

	for (int n = 0; n < numNeurons; n++)
	{
		for (int m = 0; m < numNeurons; m++)
		{
			if (n != m)
			{
				if (n < numExcit)
				{
					if (dist(rd) < .15)
					{
						d = intDist(rd);
						w = dist(rd) * 150;
						Edge e(n, m, d, w);
						h_edges.push_back(e);
					}
				}
				else
				{
					if (dist(rd) < .3)
					{
						d = intDist(rd);
						w = dist(rd) * -400;
						Edge e(n, m, d, w);
						h_edges.push_back(e);
					}
				}
			}
		}
	}

	int numEdges = h_edges.size();

	/* CUDA Memory Functions */

	hipMalloc((void**)&d_neurons, numNeurons * sizeof(Neuron));
	hipMalloc((void**)&d_I, numNeurons * maxDelay * sizeof(float));
	hipMalloc((void**)&d_driven, numNeurons * sizeof(float));
	hipMalloc((void**)&d_cf, numNeurons * sizeof(bool));
	hipMalloc((void**)&d_v, numNeurons * sizeof(float));
	hipMalloc((void**)&d_inputs, numNeurons * sizeof(float));
	hipMalloc((void**)&d_drivenZero, numNeurons * sizeof(float));


	hipMalloc((void**)&d_edges, numEdges * sizeof(Edge));


	hipMemcpy(d_neurons, h_neurons, numNeurons * sizeof(Neuron), hipMemcpyHostToDevice);
	hipMemcpy(d_I, h_I, numNeurons * maxDelay * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_driven, h_driven, numNeurons * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_drivenZero, h_drivenZero, numNeurons * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_edges, h_edges.data(), numEdges * sizeof(Edge), hipMemcpyHostToDevice);


	/* Run Simulation */

	for (int t = 0; t < equilizationTime; t++)
	{
		/* Run Timesteps, No Communication */
		NeuronTimestep << <(numNeurons + numThreads - 1) / numThreads, numThreads >> >(
			numNeurons,
			numExcit,
			d_neurons,
			d_I,
			d_cf,
			d_drivenZero,
			t,
			maxDelay,
			d_v,
			d_inputs);
	}

	for (int t = 0; t < transientTime; t++)
	{
		/* Run Timesteps, Communication, No Writing */
		NeuronTimestep << <(numNeurons + numThreads - 1) / numThreads, numThreads >> >(
			numNeurons,
			numExcit,
			d_neurons,
			d_I,
			d_cf,
			d_drivenZero,
			t,
			maxDelay,
			d_v,
			d_inputs);

		CommunicationPhase << <(numEdges + numThreads - 1) / numThreads, numThreads >> >(
			numEdges,
			d_cf,
			d_edges,
			d_I,
			t,
			maxDelay);

	}

	for (int t = 0; t < T; t++)
	{
		/* Run Timesteps, Communication, Write Results*/
		NeuronTimestep << <(numNeurons + numThreads - 1) / numThreads, numThreads >> >(
			numNeurons,
			numExcit,
			d_neurons,
			d_I,
			d_cf,
			d_driven,
			t,
			maxDelay,
			d_v,
			d_inputs);

		CommunicationPhase << <(numEdges + numThreads - 1) / numThreads, numThreads >> >(
			numEdges,
			d_cf,
			d_edges,
			d_I,
			t,
			maxDelay);

		hipMemcpy(SpikeTrainYard[t], d_cf, numNeurons * sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(VoltageTrace[t], d_v, numNeurons * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(InputTrace[t], d_inputs, numNeurons * sizeof(float), hipMemcpyDeviceToHost);
	}


	/* Analyzing Run */

	std::vector<std::vector<int>> Firings;

	for (int t = 0; t < T; t++)
	{
		//std::cout << VoltageTrace[t][0] << "," << InputTrace[t][0] <<  std::endl;
		for (int n = 0; n < numNeurons; n++)
		{
			if (SpikeTrainYard[t][n] == true)
			{
				std::vector<int> v;
				v.push_back(t);
				v.push_back(n);
				Firings.push_back(v);
			}
		}
	}

	std::cout << "There were " << Firings.size() << " firings." << std::endl;


	/* Clean Up Code */

	hipDeviceReset();

	for (int t = 0; t < T; t++)
	{
		delete[] SpikeTrainYard[t];
		delete[] VoltageTrace[t];
		delete[] InputTrace[t];
	}

	delete[] h_neurons; delete[] h_I; delete[] h_cf; delete[] SpikeTrainYard; delete[] h_driven;
	delete[] VoltageTrace;
	delete[] InputTrace;
	return 0;
}
